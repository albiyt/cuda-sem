#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {

      size_t limit = 0;

      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("cudaLimitStackSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
      printf("cudaLimitPrintfFifoSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
      printf("cudaLimitMallocHeapSize: %u\n", (unsigned)limit);

      limit = 9999;
      
      hipDeviceSetLimit(hipLimitStackSize, limit);
      hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);
      hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

      limit = 0;

      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("New cudaLimitStackSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
      printf("New cudaLimitPrintfFifoSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
      printf("New cudaLimitMallocHeapSize: %u\n", (unsigned)limit);

      return 0;
}
