/* From: http://llpanorama.wordpress.com/2008/05/21/my-first-cuda-program/ */
#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 

#define N 10

__global__ void square_array(int* a) { 
      int idx = blockIdx.x * blockDim.x + threadIdx.x; 
      if (idx < N) a[idx] = a[idx] * a[idx]; 
}

int main(void) { 
      int* hostptr, *devptr;
      int i;

      size_t nbytes = N * sizeof(int); 

      dim3 grid = {1, 1, 1};
      dim3 block = {1, 1, 1};

      int nthreads = 4; 
      int nblocks = N/nthreads + !!(N % nthreads);

      grid.x = nblocks;
      block.x = nthreads;

      hostptr = (int*) malloc(nbytes);
      hipMalloc(&devptr, nbytes);

      for (i = 0; i != N; ++i) 
            hostptr[i] = (int)i;

      hipMemcpy(devptr, hostptr, nbytes, hipMemcpyHostToDevice); 

      square_array<<<grid, block>>>(devptr); 

      hipMemcpy(hostptr, devptr, nbytes, hipMemcpyDeviceToHost); 

      for (i = 0; i != N; ++i) 
            printf("%d %d\n", i, hostptr[i]);

      hipFree(devptr); 
      free(hostptr);
}

