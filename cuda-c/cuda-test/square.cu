#include "hip/hip_runtime.h"
/* From: http://llpanorama.wordpress.com/2008/05/21/my-first-cuda-program/ */
#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 

__global__ void square_array(float *a, int N) { 
      int idx = blockIdx * blockDim + threadIdx; 
      if (idx < N) a[idx] = a[idx] * a[idx]; 
}

int main(void) { 
      float *hostptr, *devptr;
      int N = 10, i;

      size_t nbytes = N * sizeof(float); 

      int nthreads = 4; 
      int nblocks = N/nthreads + !!(N % nthreads);

      hostptr = (float*) malloc(nbytes);
      devptr = (float*) malloc(nbytes);
      hipMalloc((void**) &devptr, nbytes);

      for (i = 0; i != N; ++i) 
            hostptr[i] = (float)i;

      hipMemcpy(devptr, hostptr, nbytes, hipMemcpyHostToDevice); 
      square_array<<<nblocks, nthreads>>>(devptr, N); 
      hipMemcpy(hostptr, devptr, nbytes, hipMemcpyDeviceToHost); 

      for (i = 0; i != N; ++i) 
            printf("%d %f\n", i, hostptr[i]);

      hipFree(devptr); 
      free(hostptr);
}

