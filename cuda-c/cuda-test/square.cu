/* From: http://llpanorama.wordpress.com/2008/05/21/my-first-cuda-program/ */
#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 

#define N 10

__global__ void square_array(int* a) { 
      int idx = blockIdx.x * blockDim.x + threadIdx.x; 
      if (idx < N) a[idx] = a[idx] * a[idx]; 
}

int main(void) { 
      int* hostptr, *devptr;
      int i;
      dim3 grid, block;

      size_t nbytes = N * sizeof(int); 

      grid.x = 1;
      grid.y = 1;
      grid.z = 1;

      block.x = 1;
      block.y = 1;
      block.z = 1;

      int nthreads = 4; 
      int nblocks = N/nthreads + !!(N % nthreads);

      grid.x = nblocks;
      block.x = nthreads;

      hostptr = (int*) malloc(nbytes);
      hipMalloc(&devptr, nbytes);

      for (i = 0; i != N; ++i) 
            hostptr[i] = (int)i;

      hipMemcpy(devptr, hostptr, nbytes, hipMemcpyHostToDevice); 

      square_array<<<grid, block>>>(devptr); 

      hipMemcpy(hostptr, devptr, nbytes, hipMemcpyDeviceToHost); 

      for (i = 0; i != N; ++i) 
            printf("%d %d\n", i, hostptr[i]);

      hipFree(devptr); 
      free(hostptr);
}
