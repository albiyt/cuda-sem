#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NELEMENTS 12
// NRUNS should be < NELEMENTS.
#define NRUNS 2
#define NBLOCKS 2
// NTHREADS_PER_BLOCK*NBLOCKS should equal NELEMENTS
#define NTHREADS_PER_BLOCK 6

__global__ void sum_kernel(int* g_odata, int* g_idata, int run) {
      __shared__ int shared[NELEMENTS];
      int i, gtid = blockIdx.x * blockDim.x + threadIdx.x;
      int tid = threadIdx.x;

      shared[tid] = g_idata[gtid];
      
      __syncthreads();

      if (tid < NTHREADS_PER_BLOCK/2) {
            shared[tid] += shared[NTHREADS_PER_BLOCK/2 + tid];
      }

      __syncthreads();

      if (tid == 0) {
            for (i = 1; i != NTHREADS_PER_BLOCK/2; ++i) {
                  shared[0] += shared[i];
            }

            g_odata[run] += shared[0];
      }
}

int main(int argc, char** argv) {
      int* d_idata, *d_odata, *h_data;
      int i;
      dim3 grid = dim3(NBLOCKS, 1, 1);
      dim3 block = dim3(NTHREADS_PER_BLOCK, 1, 1);

      // Use a different stream for every run.
      hipStream_t streams[NRUNS];

      h_data = (int*)malloc(NELEMENTS * sizeof(int));

      printf("INPUT: ");
      for(i = 0; i != NELEMENTS; ++i) {

            h_data[i] = (11 + i * i) % 7;
            printf(" %d ", h_data[i]);
      }
      printf("\n");

      hipMalloc(&d_idata, NELEMENTS * sizeof(int));
      hipMalloc(&d_odata, NRUNS * sizeof(int));

      hipMemcpy(d_idata, h_data, NELEMENTS * sizeof(int), hipMemcpyHostToDevice);

      // TODO: cudaMemset()?
      // Zero d_odata.
      for (i = 0; i != NRUNS; ++i) {
            h_data[i] = 0;
      }
      hipMemcpy(d_odata, h_data, NRUNS * sizeof(int), hipMemcpyHostToDevice);
      
      printf("Launching %d blocks of %d threads each " 
             "to asychronously sum the list above %d times.\n", 
             NBLOCKS, NTHREADS_PER_BLOCK, NRUNS);

      for (i = 0; i != NRUNS; ++i) {
            hipStreamCreate(&streams[i]);
            sum_kernel<<< grid, block, NELEMENTS * sizeof(int), streams[i] >>>
                  (d_odata, d_idata, i);
      }
      hipDeviceSynchronize();

      hipMemcpyAsync(h_data, d_odata, NRUNS * sizeof(int), hipMemcpyDeviceToHost, streams[0]);

      hipStreamSynchronize(streams[0]);
      hipDeviceSynchronize();

      printf("OUTPUT: ");
      for(i = 0; i != NRUNS; ++i) {
            hipStreamDestroy(streams[i]);
            printf(" %d ", h_data[i]);
      }
      printf("\n");

      free(h_data);
      hipFree(d_idata);
      hipFree(d_odata);
}
