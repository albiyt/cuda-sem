#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {
      int i;

      for (i = 0; i != 64; ++i) {
            printf("%d: %s\n", i, hipGetErrorString((hipError_t)i));
      }

      printf("%d: %s\n", 127, hipGetErrorString((hipError_t)127));
      printf("%d: %s\n", 10000, hipGetErrorString((hipError_t)10000));
}
