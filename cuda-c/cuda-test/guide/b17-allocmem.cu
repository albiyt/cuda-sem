// From Appendix B.17 of the CUDA-C Programming Guide.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define NUM_BLOCKS 20

__device__ int* dataptr[NUM_BLOCKS]; // Per-block pointer

__global__ void allocmem() {
      // Only the first thread in the block does the allocation
      // since we want only one allocation per block.
      if (threadIdx.x == 0)
            dataptr[blockIdx.x] = (int*)malloc(blockDim.x * 4);
      __syncthreads();
      // Check for failure
      if (dataptr[blockIdx.x] == NULL)
            return;
      // Zero the data with all threads in parallel
      dataptr[blockIdx.x][threadIdx.x] = 0;
}

// Simple example: store thread ID into each element
__global__ void usemem() {
      int* ptr = dataptr[blockIdx.x];
      if (ptr != NULL)
            ptr[threadIdx.x] += threadIdx.x;
}

// Print the content of the buffer before freeing it
__global__ void freemem() {
      int* ptr = dataptr[blockIdx.x];
      if (ptr != NULL)
            printf("Block %d, Thread %d: final value = %d\n",
                        blockIdx.x, threadIdx.x, ptr[threadIdx.x]);
      // Only free from one thread!
      if (threadIdx.x == 0)
            free(ptr);
}

int main() {
      hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
      // Allocate memory
      allocmem<<< NUM_BLOCKS, 10 >>>();
      // Use memory
      usemem<<< NUM_BLOCKS, 10 >>>();
      usemem<<< NUM_BLOCKS, 10 >>>();
      usemem<<< NUM_BLOCKS, 10 >>>();
      // Free memory
      freemem<<< NUM_BLOCKS, 10 >>>();
      hipDeviceSynchronize();
      return 0;
}

