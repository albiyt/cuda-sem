// From Appendix B.16 of the CUDA-C Programming Guide.

#include "stdio.h"
#include "hip/hip_runtime.h"

__global__ void helloCUDA(float f) {
      printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

int main() {
      helloCUDA<<<1, 5>>>(1.2345f);
      hipDeviceReset();
      return 0;
}

